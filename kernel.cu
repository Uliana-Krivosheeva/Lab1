#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


#define BLOCK_SIZE 16
using namespace std;

__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			sum += a[row * n + i] * b[i * k + col];
		}
		c[row * k + col] = sum;
	}
}
 
void cpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k) {
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < k; j++)
		{
			int tmp = 0;
			for (int h = 0; h < n; h++)
			{
				tmp += a[i * n + h] * b[h * k + j];
			}
			c[i * k + j] = tmp;
		}
	}
}

int main(int argc, char const* argv[])
{
	// ������ ������� ������
	int m, n, k;
	printf("please type in m n and k\n");
	scanf("%d %d %d", &m, &n, &k);


	// �������� ������ �� �����
	int* h_a, *h_b, *h_c, *h_res; // ��������� ������� ����� ��� ���������� ������������ �� �����, � �� ����� ��� � 3� ����� ������������ ��������� � �������
	hipHostMalloc((void**)&h_a, sizeof(int) * m * n);
	hipHostMalloc((void**)&h_b, sizeof(int) * n * k);
	hipHostMalloc((void**)&h_c, sizeof(int) * m * k);
	hipHostMalloc((void**)&h_res, sizeof(int) * m * k);
	float time_cpu, time_gpu;

	// ������� ��� �������� ������� ������ ������������ �� �������
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//������ ������� �������
	hipEventRecord(start, 0);

	// �������� ������ �� �������
	int* d_a, *d_b, *d_c;
	hipMalloc((void**)&d_a, sizeof(int) * m * n);
	hipMalloc((void**)&d_b, sizeof(int) * n * k);
	hipMalloc((void**)&d_c, sizeof(int) * m * k);

	// �������� �������� ������� � ����� �� ������
	hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

	// ���������� ������ ����� � �����
	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// ��������� ����
	gpu_matrix_mult << < dimGrid, dimBlock >> > (d_a, d_b, d_c, m, n, k);

	// �������� ��������� � ������� �� ����
	hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	// ����� ������� �������
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// ������� ����� ������ �� �������
	hipEventElapsedTime(&time_gpu, start, stop);


	// ������ ������� �������
	auto begin = chrono::steady_clock::now();

	// ��������� ������������ �� �����
	cpu_matrix_mult(h_a, h_b, h_res, m, n, k);

	// ����� ������� �������
	auto end = chrono::steady_clock::now();

	// ������� ����� ������ �� �����
	auto elapsed_ms = chrono::duration_cast<chrono::milliseconds>(end - begin);
	float time_cpu = elapsed_ms.count();


	printf("CPU time: %f ms.\n\n", time_cpu);
	printf("GPU time: %f ms.\n\n", time_gpu);


	//������� ������
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	hipHostFree(h_res);

	return 0;
}




